#include "hip/hip_runtime.h"
﻿#include <torch/extension.h>

__global__ void cudaLinear(float* X, float* W, float* B, float* Y, int M, int N, int K)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M)
    {
        for (int j = 0; j < K; j++)
        {
            float sum = 0.0f;

            for (int k = 0; k < N; k++)
            {
                sum += X[row * N + k] * W[k * K + j];
            }

            Y[row * K + j] = sum + B[j];
        }
    }
}

// Функция для вызова из Python
void gpu_linear(torch::Tensor input, torch::Tensor weights, torch::Tensor bias, torch::Tensor output) {

    int M = input.size(0); //строки X и Y
    int N = input.size(1); //столбики X и строки W
    int K = weights.size(1); //столбики W и Y

    int threadsPerBlock = 256;
    int blocksPerGrid = (M + threadsPerBlock - 1) / threadsPerBlock;

    cudaLinear<<<blocksPerGrid, threadsPerBlock>>>(input.data_ptr<float>(), weights.data_ptr<float>(), bias.data_ptr<float>(), output.data_ptr<float>(), M, N, K);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
	{
        throw std::runtime_error(hipGetErrorString(err));
    }
}

PYBIND11_MODULE(linear, m)
{
    m.def("gpu_linear", &gpu_linear, "Linear Layer Calculation");
}