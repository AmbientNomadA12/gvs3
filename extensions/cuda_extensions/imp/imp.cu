#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void GPUimplementation(float *a, float *b, float *result, int n) {
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if (i < n) {
        sdata[tid] = a[i] * b[i];
    } else {
        sdata[tid] = 0.0f;
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(result, sdata[0]);
    }
}

// Функция для вызова из Python
void gpu_sum(torch::Tensor a, torch::Tensor b, torch::Tensor result) {
    int n = a.size(0);

    // Параметры блока и сетки
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Вызов CUDA-ядра
    GPUimplementation<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(a.data_ptr<float>(), b.data_ptr<float>(), result.data_ptr<float>(), n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}

// Регистрация функции в модуле
PYBIND11_MODULE(imp, m) {
    m.def("gpu_sum", &gpu_sum, "Sum of two tensors");
}
